
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define N 64
#define TPB 32
#define ARRAY_SIZE 10000

__global__ void saxpy_kernel(float* x, float* y, const float a) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;    
    if(i < ARRAY_SIZE) {
        y[i] += a*x[i];
    }
}

__global__ void compare(float *x, float *y) {
    __shared__ bool b; 
    if (threadIdx.x == 0) {
        b = false;
    }
    
    __syncthreads();
    
    int i = blockIdx.x*blockDim.x + threadIdx.x;    
    if (i < ARRAY_SIZE && abs(x[i] - y[i]) > 1e-6) {
        printf("Mismatch %e, %e \n", x[i], y[i]);
        b = true;
    }
    
    __syncthreads();
    
    if (threadIdx.x == 0) {
        if (b)
            printf("Mismatch in block %d \n", blockIdx.x);
        else 
            printf("Block %d correct\n", blockIdx.x);
    }
}

void saxpy_cpu(float* x, float* y, const float a) {
    for (int i = 0; i < ARRAY_SIZE; i ++) {
        y[i] += a*x[i];    
    }
}

void initialize_array(float *x, float *y, const int size) {
    for (int i = 0; i < size; i++) {
        x[i] = (float)rand()/RAND_MAX; 
        y[i] = (float)rand()/RAND_MAX;
    }
}


int main() {
    float *xd, *yd = NULL;
    int arraySize = sizeof(float)*ARRAY_SIZE;    
    float *x = (float*)malloc(arraySize);
    float *y = (float*)malloc(arraySize);
    timeval *gpu_t1 = (timeval*) malloc(sizeof(timeval));   
    timeval *gpu_t2 = (timeval*) malloc(sizeof(timeval));   
    timeval *cpu_t1 = (timeval*) malloc(sizeof(timeval));   
    timeval *cpu_t2 = (timeval*) malloc(sizeof(timeval));   

    initialize_array(x, y, ARRAY_SIZE);
    
    hipMalloc(&xd, arraySize);
    hipMalloc(&yd, arraySize);
    hipMemcpy(xd, x, arraySize, hipMemcpyDefault);
    hipMemcpy(yd, y, arraySize, hipMemcpyDefault); 
    
    printf("Computing SAXPY on the GPU...\n");
    gettimeofday(gpu_t1, NULL);
    saxpy_kernel<<<(ARRAY_SIZE + 255)/256,256>>>(xd, yd, 5);
    hipDeviceSynchronize();
    gettimeofday(gpu_t2, NULL);
    printf("GPU time: %e seconds \n", ((gpu_t2->tv_sec + gpu_t2->tv_usec/1e6) - (gpu_t1->tv_sec + gpu_t1->tv_usec/1e6)));
    printf("computing SAXPY on the CPU...\n");
    
    gettimeofday(cpu_t1, NULL);
    saxpy_cpu(x, y, 5);
    gettimeofday(cpu_t2, NULL);
    printf("CPU time: %e seconds \n", ((cpu_t2->tv_sec + cpu_t2->tv_usec/1e6) - (cpu_t1->tv_sec + cpu_t1->tv_usec/1e6)));
    
    hipMemcpy(xd, y, arraySize, hipMemcpyDefault);
    
    printf("Comparing the output for each implementation...\n");
    compare<<<(ARRAY_SIZE + 255)/256,256>>>(xd, yd);
    hipDeviceSynchronize();

    free(x);
    free(y);
    free(gpu_t1);
    free(gpu_t2);
    free(cpu_t1);
    free(cpu_t2);
    return 0;
}
